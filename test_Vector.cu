#include "hip/hip_runtime.h"
#include "src/Vector.cu"

/* Init the vector object in the device */
template <int Size, typename T>
__global__ void init_vector_seq(Vector<Size,T> *ptr) {
    for (int i=0; i<Size; i++) {
        (*ptr)[i] = i;
    }
}

template <int Size, typename T>
__global__ 
void operate_on_vectors_device(Vector<Size,T> *v1, Vector<Size,T> *v2) {
    auto sum = 2*(3+*v1) + (1+(*v2));
    print_vector(&sum);
}


int main( int argc, char **argv )
{
    // Test vector device
    Vector<10> *vector_ptr;
    cuda_alloc_vector(&vector_ptr);
    print_vector_kernel<<<1,1>>>(vector_ptr);
    hipFree(vector_ptr);

    // Test vector device
    cuda_alloc_vector(&vector_ptr);
    init_vector_seq<<<1,1>>>(vector_ptr);
    print_vector_kernel<<<1,1>>>(vector_ptr);
    hipFree(vector_ptr);

    // Test vector operations
    Vector<> *vector_1;
    cuda_alloc_vector(&vector_1);
    Vector<> *vector_2;
    cuda_alloc_vector(&vector_2);


    operate_on_vectors_device<<<1,1>>>(vector_1, vector_2);
    hipFree(vector_1);
    hipFree(vector_2);
}