#include "hip/hip_runtime.h"
/* 
## Clase `ParticleSystem`

Esta clase está diseñada para manejarse desde el Host(CPU), lo importante es que 
contiene un `thrust::device_vector` de partículas, por lo que estas viven 
completamente en el GPU y de ahí se operan. A su vez, el `Container` forma parte 
de un `device_obj`, por lo que reside también completamente en el device.  El 
`kernel` es un integrador muy simple donde cada partícula tiene su propio hilo. 
Falta algo para calcular la fuerza, esto probablemente se podrá hacer con otro 
kernel. 
*/

#include <thrust/device_vector.h>
#include <thrust/random.h>

#include "Particle.cu"
#include "Vector.cu"
#include "Container.cu"
#include "device_obj.cu"



__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
            __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

template <typename ParticleT>
__global__                                                       
void init_kernel(ParticleT *particles, int n) {
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n){
        
          thrust::default_random_engine rng(index*1000 + index*index);
          rng.discard(index);

          // create a mapping from random numbers to [0,1)
          thrust::normal_distribution<double> dist(0, 1);

          // Create a random motion vector
          using vector_type = typename ParticleT::vector_type;
          vector_type delta;

          for (int i=0; i<delta.dimensions; i++) {
              float rnd_value = dist(rng);
              delta[i] = rnd_value;
          }

        particles[index].position = delta;
    }
}

template <typename ParticleT, typename ContainerT>
__global__ 
void force_kernel(ParticleT *particles, int n_particles, ContainerT *box) {
    unsigned int row = blockIdx.x;
    unsigned int column = blockIdx.y*blockDim.y + threadIdx.x;

    // Reset the forces
    if(column == 0) {
        particles[row].force = 0.;
    }

    __syncthreads();

    if( column > row && column < n_particles ){
        
        auto force = particles[row]
                        .force_law(&particles[column], box);
        
        for( int i=0; i<force.dimensions; ++i ){
            atomicAddDouble( &particles[row].force[i], force[i] );
            atomicAddDouble( &particles[column].force[i], -force[i] );
        }  
    }
}

template <typename ParticleT, typename ContainerT>
__global__                                                       
void first_half_kernel(ParticleT *particles, int n_particles, ContainerT *box, double dt) {
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n_particles){
        
          // r(t + dt) = r(t) + v(t)*dt + 1/2*f(t)*dt^2
          auto particle = particles[index];
          auto dr = particle.velocity*dt + 0.5*particle.force*dt*dt;
          auto new_pos = particle.position + dr;

          particles[index].position = (*box).apply_boundary_conditions(new_pos); 

          // v(t + 1/2*dt) = v(t) + 1/2*f(t)*dt
          auto dv = 0.5*particle.force*dt;
          //print_vector( &dv );
          particles[index].velocity += 0.5*particle.force*dt;
    }
}            

template <typename ParticleT, typename ContainerT>
__global__                                                       
void second_half_kernel(ParticleT *particles, int n_particles, ContainerT *box, double dt)  {
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n_particles){
        
          // v(t + dt) = v(t + 1/2*dt) + 1/2*f(t + dt)*dt
          auto particle = particles[index];
          particles[index].velocity += 0.5*particle.force*dt;
    }
}            

template< typename ParticleT=Particle<>, typename ContainerT=PeriodicBoundaryBox<> >
class ParticleSystem
{
    unsigned int n_particles;
    thrust::device_vector< ParticleT > particles;
    device_obj< ContainerT > box;

  public:
    
    using particle_type = ParticleT;
    using container_type = ContainerT;
    using vector_type = typename ParticleT::vector_type;
    static constexpr int dimensions = ParticleT::dimensions;
    
    ParticleSystem(unsigned int n, double numeric_density) 
        : n_particles{n},
          particles{thrust::device_vector<ParticleT>(n)},
          box{pow(n/numeric_density, 1./dimensions)} {};

    void integrator(double dt) { /*
        * Implementation of a velocity Vertlet integrator.
        * See: http://www.pages.drexel.edu/~cfa22/msim/node23.html#sec:nmni
        * 
        * This integrator gives a lower error O(dt^4) and more stability than
        * the standard forward integration (x(t+dt) += v*dt + 1/2 * f * dt^2)
        * by looking at more timesteps (t, t+dt) AND (t-dt), but in order to 
        * improve memory usage, the integration is done in two steps.
        */

        // r(t + dt) = r(t) + v(t)*dt + 1/2*f(t)*dt^2
        // v(t + 1/2*dt) = v(t) + 1/2*f(t)*dt
        first_half_step(dt);

        // r(t + dt)  -->  f(t + dt)
        update_forces();

        // v(t + dt) = v(t + 1/2*dt) + 1/2*f(t + dt)*dt
        second_half_step(dt);
    }

    void first_half_step(double dt) {
        
        ParticleT* particles_ptr = thrust::raw_pointer_cast(particles.data());
        ContainerT* box_ptr = box.device_ptr();

        unsigned int block_size = 1024;
        unsigned int grid_size = n_particles / block_size + 1;
        
        first_half_kernel<<<grid_size,block_size>>>(particles_ptr, n_particles, box_ptr, dt);
    }

    void second_half_step(double dt) {
        
        ParticleT* particles_ptr = thrust::raw_pointer_cast(particles.data());
        ContainerT* box_ptr = box.device_ptr();

        unsigned int block_size = 1024;
        unsigned int grid_size = n_particles / block_size + 1;
        
        second_half_kernel<<<grid_size,block_size>>>(particles_ptr, n_particles, box_ptr, dt);
    }

    void update_forces() {
        // As we cannot send device vectors to the kernel (as device_vector is at
        // the end of the day a GPU structure abstraction in CPU) we have to get the
        // pointer in GPU memory in order for the kernel to know where to start 
        // reading the particle array from.
        
        ParticleT* particles_ptr = thrust::raw_pointer_cast(particles.data());
        ContainerT* box_ptr = box.device_ptr();
      
        unsigned int block_size = 1024;
        
        // Esta wea si funciona 
        dim3 grid_size( 
                n_particles,
                n_particles / block_size + ( n_particles % block_size == 0 ? 0:1 ) 
             );  
        
        // Launch the kernel! As you can see we are not copying memory from CPU to GPU
        // as you would normally do with hipMemcpy(), as we don't need to! The
        // vectors live in GPU already so we just need to know where they start (GPU
        // pointer) and pass it to the kernel. No need to copy back, we can read from
        // the device vector with the ::operator[]() i.e. positions[2] and that would
        // do all the memory copying for us!

        // Update forces
        
        force_kernel<<<grid_size,block_size>>>(particles_ptr, n_particles, box_ptr);
    }
 
    void simulation_step(double dt) {
        integrator(dt);
    }
    
    void simulation_init() {
        
        ParticleT* particles_ptr = thrust::raw_pointer_cast(particles.data());

        unsigned int block_size = 1024;
        unsigned int grid_size = n_particles / block_size + 1;
        
        init_kernel<<<grid_size,block_size>>>(particles_ptr, n_particles);
    }

    void print() {
        printf("Container: \n\t");

        box.get();
        print_container(box.raw_ptr());

        printf("\n");
        
        thrust::host_vector<ParticleT> p(particles);

        printf("Particles: \n");
        for (int i=0; i<n_particles; i++) {
            printf("%d:\t", i);
            print_particle( &(p[i]) );
            printf("\n");
        }

    }
};