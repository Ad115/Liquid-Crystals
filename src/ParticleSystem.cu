#include "hip/hip_runtime.h"
/* 
## Clase `ParticleSystem`

Esta clase está diseñada para manejarse desde el Host(CPU), lo importante es que 
contiene un `thrust::device_vector` de partículas, por lo que estas viven 
completamente en el GPU y de ahí se operan. A su vez, el `Container` forma parte 
de un `device_obj`, por lo que reside también completamente en el device.  El 
`kernel` es un integrador muy simple donde cada partícula tiene su propio hilo. 
Falta algo para calcular la fuerza, esto probablemente se podrá hacer con otro 
kernel. 
*/

#include <thrust/device_vector.h>
#include <thrust/random.h>

#include "Particle.cu"
#include "Vector.cu"
#include "Container.cu"
#include "device_obj.cu"


// seed a random number generator
/*
// This is the kernel that is launched from CPU and GPU runs it for each cell
template <typename VectorT>
__global__ 
void integrator_kernel(Particle<VectorT> *particles, int n, int step) {
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
   

    // --- First half step ---
    map_to_particles([dt, &box=container()](ParticleClass& p){

            // r(t+dt) = r(t) + v(t)*dt + 1/2*f*dt^2
            p.position = p.position + dt * p.velocity + 1/2.*dt*dt*p.force;
            p.position = box.apply_boundary_conditions(p.position);

            // v(t+dt/2) = v(t)+f(t)/2*dt
            p.velocity = p.velocity + dt/2.*p.force;
        }
    );
    
    // r(t + dt) --> f(t + dt)
    
    // First zero out the forces
    for(auto& p : particles){ p.force = 0 * p.force; }

    for(int i=0; i<particles.size()-1; i++) {
        for(int j=i+1; j<particles.size(); j++) {
            
            Vector force = particles[i].force_law(particles[j], container());

            particles[i].force += force;
            particles[j].force -= force;
        }
    }
    
    // --- Second half step ---
    map_to_particles([dt](ParticleClass& p){

            // v(t+dt) = v(t+dt/2)+f(t+dt)/2*dt
            p.velocity = p.velocity + dt/2.*p.force;
        }
    );


}
*/
__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
            __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}




// This is the kernel that is launched from CPU and GPU runs it for each cell
template <typename ParticleT, typename ContainerT>
__global__ 
void integrator_kernel(ParticleT *particles,  ContainerT *box, unsigned int n, int step) {
    unsigned int row = gridDim.x;
    unsigned int column = blockIdx.y * blockDim.y + threadIdx.x;

    if( column > row && column < n ){
        auto force = particles[row].force_law(&particles[column], box);
        print_vector( &force );
        for( int i=0; i<force.dimensions; ++i ){
            atomicAddDouble( &particles[row].force[i], force[i] );
            atomicAddDouble( &particles[column].force[i], - force[i] );
        }  
    }
}
                                                      
template <typename ParticleT>
__global__                                                       
void init_kernel(ParticleT *particles, int n) {
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n){
        
          thrust::default_random_engine rng(index);
          rng.discard(index);

          // create a mapping from random numbers to [0,1)
          thrust::normal_distribution<double> dist(0, 1);

          // Create a random motion vector
          using vector_type = typename ParticleT::vector_type;
          vector_type delta;

          for (int i=0; i<delta.dimensions; i++) {
              float rnd_value = dist(rng);
              delta[i] = rnd_value;
          }

        particles[index].position = delta;
    }
}                                                      

template< typename ParticleT=Particle<>, typename ContainerT=PeriodicBoundaryBox<> >
class ParticleSystem
{
    unsigned int n_particles;
    thrust::device_vector< ParticleT > particles;
    device_obj< ContainerT > box;

  public:
    
    using particle_type = ParticleT;
    using container_type = ContainerT;
    using vector_type = typename ParticleT::vector_type;
    static constexpr int dimensions = ParticleT::dimensions;
    
    ParticleSystem(unsigned int n, double numeric_density) 
        : n_particles{n},
          particles{thrust::device_vector<ParticleT>(n)},
          box{pow(n/numeric_density, 1./dimensions)} {};

    void simulation_step(int step) {
        // As we cannot send device vectors to the kernel (as device_vector is at
        // the end of the day a GPU structure abstraction in CPU) we have to get the
        // pointer in GPU memory in order for the kernel to know where to start 
        // reading the particle array from.
        
        ParticleT* particles_ptr = thrust::raw_pointer_cast(particles.data());
        ContainerT* box_ptr = box.device_ptr();
      
        /* This is the way I structured my blocks and threads. I fixed the amount of
         * threads per block to 1024. So to get the amount of blocks we just get the
         * total number of elements in positions and divide it by 1024. We add one in
         * case the division leaves remainder.
         *
         * ┌──────────────────────grid─┬of─blocks─────────────────┬──────────
         * │     block_of_threads      │     block_of_threads     │  
         * │ ┌───┬───┬───────┬──────┐  │ ┌───┬───┬───────┬──────┐ │
         * │ │ 0 │ 1 │ [...] │ 1023 │  │ │ 0 │ 1 │ [...] │ 1023 │ │   ...
         * │ └───┴───┴───────┴──────┘  │ └───┴───┴───────┴──────┘ │
         * └───────────────────────────┴──────────────────────────┴──────────
         */
        
        unsigned int block_size = 1024;
        dim3 grid_size( n_particles, n_particles / block_size + ( n_particles % block_size == 0 ? 0:1 ) );  
        
        // Launch the kernel! As you can see we are not copying memory from CPU to GPU
        // as you would normally do with hipMemcpy(), as we don't need to! The
        // vectors live in GPU already so we just need to know where they start (GPU
        // pointer) and pass it to the kernel. No need to copy back, we can read from
        // the device vector with the ::operator[]() i.e. positions[2] and that would
        // do all the memory copying for us!
        
        integrator_kernel<<<grid_size,block_size>>>(particles_ptr, box_ptr, n_particles, step);
    }
    
    void simulation_init() {
        
        ParticleT* particles_ptr = thrust::raw_pointer_cast(particles.data());

        unsigned int block_size = 1024;
        unsigned int grid_size = n_particles / block_size + 1;
        
        init_kernel<<<grid_size,block_size>>>(particles_ptr, n_particles);
    }

    void print() {
        printf("Container: \n\t");

        box.get();
        print_container(box.raw_ptr());

        printf("\n");
        
        thrust::host_vector<ParticleT> p(particles);

        printf("Particles: \n");
        for (int i=0; i<(n_particles-1); i++) {
            printf("\t");
            print_particle( &(p[i]) );
            printf("\n");
        }

    }
};
