#include "hip/hip_runtime.h"
/* 
## Clase `ParticleSystem`

Esta clase está diseñada para manejarse desde el Host(CPU), lo importante es que 
contiene un `thrust::device_vector` de partículas, por lo que estas viven 
completamente en el GPU y de ahí se operan. A su vez, el `Container` forma parte 
de un `device_obj`, por lo que reside también completamente en el device.  El 
`kernel` es un integrador muy simple donde cada partícula tiene su propio hilo. 
Falta algo para calcular la fuerza, esto probablemente se podrá hacer con otro 
kernel. 
*/

#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/transform_reduce.h>
#include <thrust/for_each.h>
#include <iostream>
#include "Particle.cu"
#include "Vector.cu"
#include "Container.cu"
#include "device_obj.cu"
#include "Transformations.cu"
#include "InitialConditions.cu"



template <typename ParticleT, typename ContainerT>
__global__ 
void force_kernel(ParticleT *particles, int n_particles, ContainerT *box) {
    unsigned int row = blockIdx.x;
    unsigned int column = blockIdx.y*blockDim.y + threadIdx.x;

    // Reset the forces
    if(column == 0) {
        particles[row].force = 0.;
    }

    __syncthreads();

    if( column > row && column < n_particles ){
        
        double cutoff_radius = 3.5;
        auto dr = box->distance_vector(
            particles[row].position, 
            particles[column].position
        );

        if ((dr * dr) < (cutoff_radius * cutoff_radius)) {
            auto force = particles[row]
                .interaction_force_with(particles[column], *box);

            for( int i=0; i<force.dimensions; ++i ){
                atomicAdd( &particles[row].force[i], force[i] );
                atomicAdd( &particles[column].force[i], -force[i] );
            }  
        }
        
    }
}

template <typename ParticleT, typename ContainerT>
__global__                                                       
void first_half_kernel(ParticleT *particles, int n_particles, ContainerT *box, double dt) {
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n_particles){
        
          // r(t + dt) = r(t) + v(t)*dt + 1/2*f(t)*dt^2
          auto particle = particles[index];
          auto dr = particle.velocity*dt + 0.5*particle.force*dt*dt;
          auto new_pos = particle.position + dr;

          particles[index].position = (*box).apply_boundary_conditions(new_pos); 

          // v(t + 1/2*dt) = v(t) + 1/2*f(t)*dt
          auto dv = 0.5*particle.force*dt;
          //print_vector( &dv );
          particles[index].velocity += 0.5*particle.force*dt;
    }
}            

template <typename ParticleT, typename ContainerT>
__global__                                                       
void second_half_kernel(ParticleT *particles, int n_particles, ContainerT *box, double dt)  {
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n_particles){
        
          // v(t + dt) = v(t + 1/2*dt) + 1/2*f(t + dt)*dt
          auto particle = particles[index];
          particles[index].velocity += 0.5*particle.force*dt;
    }
}            

template< typename ParticleT=Particle<>, typename ContainerT=EmptySpace<> >
class ParticleSystem
{

 public:
    thrust::device_vector< ParticleT > particles;
    device_obj< ContainerT > box;

    unsigned int n_particles;
    
    using particle_type = ParticleT;
    using container_type = ContainerT;
    using vector_type = typename ParticleT::vector_type;
    static constexpr int dimensions = ParticleT::dimensions;

    ParticleSystem(unsigned int n, double numeric_density) 
        : n_particles{n},
          particles{thrust::device_vector<ParticleT>(n)},
          box{pow(n/numeric_density, 1./dimensions)} {};

    template <typename MeasureFn>
    double measure_particles(MeasureFn measure_fn) {
        return 
            thrust::transform_reduce(
                    particles.begin(), particles.end(), 
                    measure_fn,
                    0.,
                    thrust::plus<double>{}
            );
    }

    template <typename ParticleFn>
    ParticleFn map_to_particles(ParticleFn particle_fn) {
        
        thrust::for_each(
            particles.begin(), particles.end(),
            particle_fn
        );

        return particle_fn;
    }

    template<typename TransformationT>
    TransformationT apply(TransformationT transformation) {
        transformation(*this);
        return transformation;
    }

    void simulation_step(double dt) {
        integrator(dt);
    }
    
    void simulation_init() {
        (*this).apply(initial_conditions{});
    }

    void integrator(double dt) { /*
        * Implementation of a velocity Vertlet integrator.
        * See: http://www.pages.drexel.edu/~cfa22/msim/node23.html#sec:nmni
        * 
        * This integrator gives a lower error O(dt^4) and more stability than
        * the standard forward integration (x(t+dt) += v*dt + 1/2 * f * dt^2)
        * by looking at more timesteps (t, t+dt) AND (t-dt), but in order to 
        * improve memory usage, the integration is done in two steps.
        */

        // r(t + dt) = r(t) + v(t)*dt + 1/2*f(t)*dt^2
        // v(t + 1/2*dt) = v(t) + 1/2*f(t)*dt
        first_half_step(dt);

        // r(t + dt)  -->  f(t + dt)
        update_forces();

        // v(t + dt) = v(t + 1/2*dt) + 1/2*f(t + dt)*dt
        second_half_step(dt);
    }

    void first_half_step(double dt) {
        
        ParticleT* particles_ptr = thrust::raw_pointer_cast(particles.data());
        ContainerT* box_ptr = box.device_ptr();

        unsigned int block_size = 1024;
        unsigned int grid_size = n_particles / block_size + 1;
        
        first_half_kernel<<<grid_size,block_size>>>(particles_ptr, n_particles, box_ptr, dt);
    }

    void second_half_step(double dt) {
        
        ParticleT* particles_ptr = thrust::raw_pointer_cast(particles.data());
        ContainerT* box_ptr = box.device_ptr();

        unsigned int block_size = 1024;
        unsigned int grid_size = n_particles / block_size + 1;
        
        second_half_kernel<<<grid_size,block_size>>>(particles_ptr, n_particles, box_ptr, dt);
    }

    void update_forces() {
        // As we cannot send device vectors to the kernel (as device_vector is at
        // the end of the day a GPU structure abstraction in CPU) we have to get the
        // pointer in GPU memory in order for the kernel to know where to start 
        // reading the particle array from.
        
        ParticleT* particles_ptr = thrust::raw_pointer_cast(particles.data());
        ContainerT* box_ptr = box.device_ptr();
      
        unsigned int block_size = 1024;
        
        // Esta wea si funciona 
        dim3 grid_size( 
                n_particles,
                n_particles / block_size + ( n_particles % block_size == 0 ? 0:1 ) 
             );  
        
        // Launch the kernel! As you can see we are not copying memory from CPU to GPU
        // as you would normally do with hipMemcpy(), as we don't need to! The
        // vectors live in GPU already so we just need to know where they start (GPU
        // pointer) and pass it to the kernel. No need to copy back, we can read from
        // the device vector with the ::operator[]() i.e. positions[2] and that would
        // do all the memory copying for us!

        // Update forces
        
        force_kernel<<<grid_size,block_size>>>(particles_ptr, n_particles, box_ptr);
    }

    void print() {
        printf("Container: \n\t");

        box.get();
        print_container(box.raw_ptr());

        printf("\n");
        
        thrust::host_vector<ParticleT> p(particles);

        printf("Particles: \n");
        for (int i=0; i<n_particles; i++) {
            printf("%d:\t", i);
            print_particle( &(p[i]) );
            printf("\n");
        }

    }

    void write_xyz(std::ostream& stream) { /*
        * Output the positions of the particles in the XYZ format.
        * The format consists in a line with the number of particles,
        * then a comment line followed by the space-separated coordinates 
        * of each particle in different lines.
        * 
        * Example (for 3 particles in the xyz diagonal):
        * 
        *   10
        *   
        *   1.0 1.0 1.0
        *   1.5 1.5 1.5
        *   2.0 2.0 2.0
        */
        
        thrust::host_vector<ParticleT> host_particles = particles;
        stream << n_particles << "\n";
        for (ParticleT p: host_particles) {
            stream << "\n";
            for (int D = 0; D < dimensions; D++)
                stream << p.position[D] << " ";
        }
        stream << std::endl;
        return;
    };
};

template<typename ParticleT, typename ContainerT>
constexpr int ParticleSystem<ParticleT, ContainerT>::dimensions;