#include "hip/hip_runtime.h"
/* 
## Clase `ParticleSystem`

Esta clase está diseñada para manejarse desde el Host(CPU), lo importante es que 
contiene un `thrust::device_vector` de partículas, por lo que estas viven 
completamente en el GPU y de ahí se operan. A su vez, el `Container` forma parte 
de un `device_obj`, por lo que reside también completamente en el device.  El 
`kernel` es un integrador muy simple donde cada partícula tiene su propio hilo. 
Falta algo para calcular la fuerza, esto probablemente se podrá hacer con otro 
kernel. 
*/

#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <iostream>
#include "Particle.cu"
#include "Vector.cu"
#include "Container.cu"
#include "device_obj.cu"
/*
class simple_cubic_lattice {
    
    unsigned dimensions;
    double L;
    int cube_length;
    int particle_idx=0;

    public:

        template< typename ParticleSystem >
        void fetch_parameters_from(const ParticleSystem& system) {
            dimensions = system.dimensions();
            L = system.container().side_length();

             // No. of particles along every side of the cube
            cube_length = ceil(pow(system.n_particles(), 1./system.dimensions()));
                  // The lowest integer such that cube_length^DIMENSIONS >= n. 
                  // Think of a cube with side cube_length where all particles 
                  // are evenly spaced on a simfunction ple grid.
        } 

        template< typename ParticleSystem>
        void operator()(ParticleSystem& system) { 

            fetch_parameters_from(system);

            using Particle = typename ParticleSystem::Particle_t;
            system.map_to_particles([this](Particle& p) { (*this).particle_fn(p); });
        }

        template< typename ParticleClass>
        void particle_fn(ParticleClass& p) {

            Vector position(dimensions);
            for (int D=0; D<dimensions; D++) {
                // Get position in a hypercube with volume = cube_length^DIMENSIONS.
                position[D] = ((int)( (particle_idx / pow(cube_length, D)) )%cube_length);
                // Rescale to a box of volume = L^DIMENSIONS
                position[D] *= (L/cube_length)*0.9; // The 0.9 factor is for safety,
                                                    // particles on the edges aren't
                                                    // too close.
            }
            p.set_position(position);
            particle_idx++;
        }
};
*/


__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
            __double_as_longlong(val + __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

template <typename ParticleT, typename ContainerT>
__global__                                                       
void init_kernel(ParticleT *particles, int n, ContainerT *box) {
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n){
        
          thrust::default_random_engine rng(index*1000 + index*index);
          rng.discard(index);

          // create a mapping from random numbers to [0,1)
          double L = (*box).side_length;
          thrust::uniform_real_distribution<double> dist(0, L);

          // Create a random motion vector
          using vector_type = typename ParticleT::vector_type;
          vector_type delta;

          for (int i=0; i<delta.dimensions; i++) {
              float rnd_value = dist(rng);
              delta[i] = rnd_value;
          }

        particles[index].position = (*box).apply_boundary_conditions(delta);
    }
}

template <typename ParticleT, typename ContainerT>
__global__ 
void force_kernel(ParticleT *particles, int n_particles, ContainerT *box) {
    unsigned int row = blockIdx.x;
    unsigned int column = blockIdx.y*blockDim.y + threadIdx.x;

    // Reset the forces
    if(column == 0) {
        particles[row].force = 0.;
    }

    __syncthreads();

    if( column > row && column < n_particles ){
        
        auto force = particles[row]
                        .force_law(&particles[column], box);
        
        for( int i=0; i<force.dimensions; ++i ){
            atomicAddDouble( &particles[row].force[i], force[i] );
            atomicAddDouble( &particles[column].force[i], -force[i] );
        }  
    }
}

template <typename ParticleT, typename ContainerT>
__global__                                                       
void first_half_kernel(ParticleT *particles, int n_particles, ContainerT *box, double dt) {
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n_particles){
        
          // r(t + dt) = r(t) + v(t)*dt + 1/2*f(t)*dt^2
          auto particle = particles[index];
          auto dr = particle.velocity*dt + 0.5*particle.force*dt*dt;
          auto new_pos = particle.position + dr;

          particles[index].position = (*box).apply_boundary_conditions(new_pos); 

          // v(t + 1/2*dt) = v(t) + 1/2*f(t)*dt
          auto dv = 0.5*particle.force*dt;
          //print_vector( &dv );
          particles[index].velocity += 0.5*particle.force*dt;
    }
}            

template <typename ParticleT, typename ContainerT>
__global__                                                       
void second_half_kernel(ParticleT *particles, int n_particles, ContainerT *box, double dt)  {
    unsigned int index = blockDim.x * blockIdx.x + threadIdx.x;
    if (index < n_particles){
        
          // v(t + dt) = v(t + 1/2*dt) + 1/2*f(t + dt)*dt
          auto particle = particles[index];
          particles[index].velocity += 0.5*particle.force*dt;
    }
}            

template< typename ParticleT=Particle<>, typename ContainerT=PeriodicBoundaryBox<> >
class ParticleSystem
{
    unsigned int n_particles;
    thrust::device_vector< ParticleT > particles;
    device_obj< ContainerT > box;

  public:
    
    using particle_type = ParticleT;
    using container_type = ContainerT;
    using vector_type = typename ParticleT::vector_type;
    static constexpr int dimensions = ParticleT::dimensions;

    ParticleSystem(unsigned int n, double numeric_density) 
        : n_particles{n},
          particles{thrust::device_vector<ParticleT>(n)},
          box{pow(n/numeric_density, 1./dimensions)} {};

    void integrator(double dt) { /*
        * Implementation of a velocity Vertlet integrator.
        * See: http://www.pages.drexel.edu/~cfa22/msim/node23.html#sec:nmni
        * 
        * This integrator gives a lower error O(dt^4) and more stability than
        * the standard forward integration (x(t+dt) += v*dt + 1/2 * f * dt^2)
        * by looking at more timesteps (t, t+dt) AND (t-dt), but in order to 
        * improve memory usage, the integration is done in two steps.
        */

        // r(t + dt) = r(t) + v(t)*dt + 1/2*f(t)*dt^2
        // v(t + 1/2*dt) = v(t) + 1/2*f(t)*dt
        first_half_step(dt);

        // r(t + dt)  -->  f(t + dt)
        update_forces();

        // v(t + dt) = v(t + 1/2*dt) + 1/2*f(t + dt)*dt
        second_half_step(dt);
    }

    void first_half_step(double dt) {
        
        ParticleT* particles_ptr = thrust::raw_pointer_cast(particles.data());
        ContainerT* box_ptr = box.device_ptr();

        unsigned int block_size = 1024;
        unsigned int grid_size = n_particles / block_size + 1;
        
        first_half_kernel<<<grid_size,block_size>>>(particles_ptr, n_particles, box_ptr, dt);
    }

    void second_half_step(double dt) {
        
        ParticleT* particles_ptr = thrust::raw_pointer_cast(particles.data());
        ContainerT* box_ptr = box.device_ptr();

        unsigned int block_size = 1024;
        unsigned int grid_size = n_particles / block_size + 1;
        
        second_half_kernel<<<grid_size,block_size>>>(particles_ptr, n_particles, box_ptr, dt);
    }

    void update_forces() {
        // As we cannot send device vectors to the kernel (as device_vector is at
        // the end of the day a GPU structure abstraction in CPU) we have to get the
        // pointer in GPU memory in order for the kernel to know where to start 
        // reading the particle array from.
        
        ParticleT* particles_ptr = thrust::raw_pointer_cast(particles.data());
        ContainerT* box_ptr = box.device_ptr();
      
        unsigned int block_size = 1024;
        
        // Esta wea si funciona 
        dim3 grid_size( 
                n_particles,
                n_particles / block_size + ( n_particles % block_size == 0 ? 0:1 ) 
             );  
        
        // Launch the kernel! As you can see we are not copying memory from CPU to GPU
        // as you would normally do with hipMemcpy(), as we don't need to! The
        // vectors live in GPU already so we just need to know where they start (GPU
        // pointer) and pass it to the kernel. No need to copy back, we can read from
        // the device vector with the ::operator[]() i.e. positions[2] and that would
        // do all the memory copying for us!

        // Update forces
        
        force_kernel<<<grid_size,block_size>>>(particles_ptr, n_particles, box_ptr);
    }
 
    void simulation_step(double dt) {
        integrator(dt);
    }
    
    void simulation_init() {
        
        ParticleT* particles_ptr = thrust::raw_pointer_cast(particles.data());
        ContainerT* box_ptr = box.device_ptr();

        unsigned int block_size = 1024;
        unsigned int grid_size = n_particles / block_size + 1;
        
        init_kernel<<<grid_size,block_size>>>(particles_ptr, n_particles, box_ptr);
    }

    void print() {
        printf("Container: \n\t");

        box.get();
        print_container(box.raw_ptr());

        printf("\n");
        
        thrust::host_vector<ParticleT> p(particles);

        printf("Particles: \n");
        for (int i=0; i<n_particles; i++) {
            printf("%d:\t", i);
            print_particle( &(p[i]) );
            printf("\n");
        }

    }

    void write_xyz(std::ostream& stream) { /*
        * Output the positions of the particles in the XYZ format.
        * The format consists in a line with the number of particles,
        * then a comment line followed by the space-separated coordinates 
        * of each particle in different lines.
        * 
        * Example (for 3 particles in the xyz diagonal):
        * 
        *   10
        *   
        *   1.0 1.0 1.0
        *   1.5 1.5 1.5
        *   2.0 2.0 2.0
        */
        
        thrust::host_vector<ParticleT> host_particles = particles;
        stream << n_particles << "\n";
        for (ParticleT p: host_particles) {
            stream << "\n";
            for (int D = 0; D < dimensions; D++)
                stream << p.position[D] << " ";
        }
        stream << std::endl;
        return;
    };
};