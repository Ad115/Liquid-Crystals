#include "hip/hip_runtime.h"
#pragma once

#include "pcuditas/gpu/gpu_array.cu"


template<class ParticleT, class EnvironmentT>
__global__ 
void update_forces_shared_kernel(
        ParticleT *particles, int n_particles, 
        EnvironmentT *env_ptr) {

    extern __shared__ ParticleT particles_sh[];
    using vector_t = typename ParticleT::vector_type;
    EnvironmentT env = (*env_ptr);

    for (int i = blockIdx.x*blockDim.x + threadIdx.x; 
         i < n_particles; 
         i += blockDim.x*gridDim.x) {
        
        auto force = vector_t::zero();
        auto self_pos = particles[i].position;

        // For every other particle
        for (int j=0; j<n_particles; j += blockDim.x) {
            // Copy to shared memory
            particles_sh[threadIdx.x] = particles[j + threadIdx.x];
            __syncthreads();

            // Reduce on block
            for(size_t k=0; k<blockDim.x; k++) {
                auto other_pos = particles[k].position;
                auto dr = env.distance_vector(other_pos, self_pos);

                auto f_ij = (i != k) ? ParticleT::force_law(dr) : vector_t::zero();
                force += f_ij;   
            }
            __syncthreads();
        }

        // Save the results
        particles[i].force = force;
    }
}

template<class ParticleT, class EnvironmentT>
void update_forces_shared(
            gpu_array<ParticleT> &particles,
            gpu_object<EnvironmentT> &env,
            unsigned int block_size = 1024,
            unsigned int threads_per_block = 32) {
        
        // Launch the kernel. As you can see we are not copying memory from CPU to GPU
        // as you would normally do with hipMemcpy(), as we don't need to! The
        // vectors live in GPU already so we just need to know where they start (GPU
        // pointer) and pass it to the kernel.

        unsigned int shared_memory_size = threads_per_block * sizeof(ParticleT);
        update_forces_shared_kernel<<<block_size, threads_per_block, shared_memory_size>>>(
            particles.gpu_pointer(), particles.size, 
            env.gpu_pointer()
        );
}


// ===   ===   ===

template<class ParticleT, class EnvironmentT>
void update_forces_naive(
        gpu_array<ParticleT> &particles,
        gpu_object<EnvironmentT> &environment) {

    using vector_t = typename ParticleT::vector_type;

    // Naïve paralellization.
    particles.for_each(
        [others=particles.gpu_pointer(), n=particles.size, 
         env_ptr=environment.gpu_pointer()] 
        __device__ (ParticleT& self, int i) {
            auto force = vector_t::zero();
            auto self_pos = self.position;
            for(int j=0; j<n; j++) {
                auto other_pos = others[j].position;
                auto dr = env_ptr->distance_vector(other_pos, self_pos);
                
                auto f_ij = (i != j) ? ParticleT::force_law(dr) : vector_t::zero();
                force += f_ij;    
            }
        
            self.force = force;
        }
    );
}


// ===  ===  === 


__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

template<class ParticleT, class EnvironmentT>
__global__ 
void update_forces_atomic_kernel(
        ParticleT *particles, int n_particles,
        EnvironmentT *env_ptr) {

    EnvironmentT env = (*env_ptr);

    for (int k = blockIdx.x*blockDim.x + threadIdx.x; 
         k < n_particles*n_particles; 
         k += blockDim.x*gridDim.x) {

        int i = k % n_particles;
        int j = k/n_particles;
        
        double cutoff_radius = 3.5;
        auto dr = env.distance_vector(
            particles[j].position,
            particles[i].position 
        );

        if (dr.magnitude() < cutoff_radius) {
            auto force = ParticleT::force_law(dr);

            for( int d=0; d<force.dimensions; ++d ){
                atomicAddDouble( &(particles[i].force[d]), force[d] );
            }  
        }
    }
}

template<class ParticleT, class EnvironmentT>
void update_forces_atomic(
        gpu_array<ParticleT> &particles,
        gpu_object<EnvironmentT> &environment) {

        using vector_t = typename ParticleT::vector_type;

        // First, reset forces
        particles.for_each([] __device__ (ParticleT& self, int i){
                self.force = vector_t::zero();
        });
        
        // Launch the kernel! As you can see we are not copying memory from CPU to GPU
        // as you would normally do with hipMemcpy(), as we don't need to! The
        // vectors live in GPU already so we just need to know where they start (GPU
        // pointer) and pass it to the kernel.

        unsigned int block_size = 1024;
        unsigned int threads_per_block = 32;
        update_forces_atomic_kernel<<<block_size,threads_per_block>>>(
            particles.gpu_pointer(), particles.size,
            environment.gpu_pointer()
        );
}

