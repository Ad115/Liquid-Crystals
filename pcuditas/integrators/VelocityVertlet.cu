#pragma once

#include "pcuditas/gpu/gpu_array.cu"
#include "pcuditas/gpu/gpu_object.cu"
#include "pcuditas/environments/EmptySpace.cu"
#include "pcuditas/interactions/LennardJones.cu"
#include "pcuditas/integrators/force_calculation/shared2.cu"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <memory>


class VelocityVertlet { /*
    * The simplest integrator (Runge-Kutta):
    *   1. update f
    *   2. x -> x + v dt;
    *   3. v -> v + f dt;
    */
    gpu_object<EmptySpace> default_environment;
public:

    VelocityVertlet() = default;

    template <class ParticleT, class EnvironmentT, class InteractionT>
    void integration_step(
            gpu_array<ParticleT> &particles, 
            gpu_object<EnvironmentT> &environment,
            InteractionT interaction,
            double dt = 0.001,
            int n_blocks = 1024,
            int threads_per_block = 32) { /*
        * Implementation of a velocity Vertlet integrator.
        * See: http://www.pages.drexel.edu/~cfa22/msim/node23.html#sec:nmni
        * 
        * This integrator gives a lower error O(dt^4) and more stability than
        * the standard forward integration (x(t+dt) += v*dt + 1/2 * f * dt^2)
        * by looking at more timesteps (t, t+dt) AND (t-dt), but in order to 
        * improve memory usage, the integration is done in two steps.
        */

        // r(t + dt) = r(t) + v(t)*dt + 1/2*f(t)*dt^2
        update_positions(particles, environment, dt);
        // v(t + 1/2*dt) = v(t) + 1/2*f(t)*dt
        update_velocities(particles, dt);

        // r(t + dt)  -->  f(t + dt)
        update_forces(particles, environment, interaction, n_blocks, threads_per_block);

        // v(t + dt) = v(t + 1/2*dt) + 1/2*f(t + dt)*dt
        update_velocities(particles, dt);
    }

    template <class ParticleT, class EnvironmentT>
    void update_positions(
            gpu_array<ParticleT> &particles, 
            gpu_object<EnvironmentT> &env,
            double dt = 0.001) {

        
        particles.for_each(
            [dt, box_ptr=env.gpu_pointer()] 
            __device__ (ParticleT &p, int i) {

                // r(t + dt) = r(t) + v(t)*dt + 1/2*f(t)*dt^2
                auto new_pos = p.position + p.velocity*dt + 0.5*p.force*dt*dt;

                p.position = box_ptr->apply_boundary_conditions(new_pos); 
            }
        );
    }

    template <class ParticleT>
    void update_velocities(gpu_array<ParticleT> &particles, double dt = 0.001) {

        particles.for_each(
            [dt] __device__ (ParticleT &p, int i) {

                // v(t + dt) = v(t) + 1/2*f(t + dt)*dt
                p.velocity = p.velocity + 0.5*p.force*dt;
            }
        );
    }
    
    template <class ParticleT, class EnvironmentT, class InteractionT>
    void update_forces(
                gpu_array<ParticleT> &particles, 
                gpu_object<EnvironmentT> &env,
                InteractionT &interaction,
                int n_blocks,
                int threads_per_block) {

        using vector_t = typename ParticleT::vector_type;

        auto update_force = 
            [particles_gpu=particles.gpu_pointer()]
            __device__
            (ParticleT &p, vector_t &force, int idx) {
                p.force = force;
        };

        
        update_forces_shared2(
            particles, interaction, vector_t::zero(), update_force,
            n_blocks, threads_per_block
        );
    }

    template <class ParticleT, class InteractionT>
    void operator()(
            gpu_array<ParticleT> &particles,
            InteractionT &interaction,
            double dt = 0.001,
            int n_blocks = 1024,
            int threads_per_block = 32) {

        integration_step(particles, default_environment, interaction, dt, n_blocks, threads_per_block);
    }

    template <class ParticleT, class EnvironmentT, class InteractionT>
    void operator()(
            gpu_array<ParticleT> &particles, 
            gpu_object<EnvironmentT> &env,
            InteractionT &interaction,
            double dt = 0.001,
            int n_blocks = 1024,
            int threads_per_block = 32) {

        integration_step(particles, env, interaction, dt, n_blocks, threads_per_block);
    }
};


