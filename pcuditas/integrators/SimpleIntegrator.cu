#pragma once

#include "pcuditas/gpu/gpu_array.cu"
#include "pcuditas/gpu/gpu_object.cu"
#include "pcuditas/environments/EmptySpace.cu"
#include "force_calculation/shared2.cu"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <memory>


class SimpleIntegrator { /*
    * The simplest integrator (Runge-Kutta):
    *   1. update f
    *   2. x -> x + v dt;
    *   3. v -> v + f dt;
    */
    gpu_object<EmptySpace> default_environment;
public:

    SimpleIntegrator() = default;

    template <class ParticleT, class EnvironmentT>
    void operator()(
            gpu_array<ParticleT> &particles, 
            gpu_object<EnvironmentT> &env,
            double dt = 0.001) {
        
        this->move(particles, dt);

        this->apply_boundary_conditions(particles, env);
    }

    template <class ParticleT, class EnvironmentT, class InteractionT>
    void operator()(
            gpu_array<ParticleT> &particles, 
            gpu_object<EnvironmentT> &env,
            InteractionT &interaction,
            double dt = 0.001) {

        this->update_forces(particles, interaction);
        
        this->move(particles, dt);

        this->apply_boundary_conditions(particles, env);
    }


    template <class ParticleT, class EnvironmentT>
    void apply_boundary_conditions(
                gpu_array<ParticleT> &particles, 
                gpu_object<EnvironmentT> &env) {

        particles.for_each(
            [env_ptr=env.gpu_pointer()] 
            __device__ 
            (ParticleT &p, size_t idx) {
                p.position = env_ptr->apply_boundary_conditions(p.position);
            }
        );
    }

    template <class ParticleT, class InteractionT>
    void update_forces(
                gpu_array<ParticleT> &particles, 
                InteractionT &interaction) {

        using vector_t = typename ParticleT::vector_type;

        auto update_force = 
            [particles_gpu=particles.gpu_pointer()]
            __device__
            (ParticleT &p, vector_t &force, int idx) {
                p.force = force;
        };

        
        update_forces_shared2(
            particles, interaction, vector_t::zero(), update_force
        );
    }

    template <class ParticleT>
    void move(
            gpu_array<ParticleT> &particles,
            double dt = 0.01) {

        particles.for_each(
            [dt] 
            __device__ (ParticleT &p, size_t i) {
                // x -> x + v dt;
                p.position += p.velocity * dt;

                // v -> v + f dt;
                p.velocity += 1/2. * p.force * dt * dt;
            }
        );
    }
};


/* -----------------------------------------------------------------------

 The following is executable documentation as described in Kevlin Henney's talk 
    "Structure and Interpretation of Test Cases" (https://youtu.be/tWn8RA_DEic)
    written using the doctest framework (https://github.com/onqtam/doctest). 

 Run with `make test`.
*/

#ifdef __TESTING__

#include "tests/doctest.h"
#include <typeinfo>   // operator typeid
#include "pcuditas/initial_conditions/random.cu"
#include "pcuditas/particles/SimpleParticle.cu"
#include "pcuditas/vectors/EuclideanVector.cu"

TEST_SUITE("Simple Integrator specification") {

    SCENARIO("No forces") {
        GIVEN("A GPU array of particles with arbitrary positions and velocities") {

            using vector_t = EuclideanVector<3>;
            using particle_t = Particle<vector_t>;
            double L = 100.;
            double V = 10.;

            auto particles = gpu_array<particle_t>(100);
            set_random_positions(particles, L);
            set_random_velocities(particles, V);

            auto snapshot = particles.copy();

            WHEN("A Simple Integrator is used to move them") {
                auto integrator = SimpleIntegrator{};
                int steps = 1000;
                for (int i=0; i<steps; i++) {
                    integrator.move(particles);
                }

                THEN("Each particle has moved along the direction of their velocity") {

                    auto are_colineal = [] (vector_t v1, vector_t v2) {
                        return (
                            (v1 * v2) / (v1.magnitude() * v2.magnitude() )
                            == doctest::Approx(1.)
                        );
                    };
                    
                    snapshot.to_cpu();
                    particles.to_cpu();
                    for (int i=0; i<particles.size; i++) {
                        auto initial_velocity = snapshot[i].velocity;
                        auto final_velocity = particles[i].velocity;

                        // Velocity didn't change
                        CHECK(initial_velocity == final_velocity);

                        auto displacement = particles[i].position - snapshot[i].position;

                        CHECK(are_colineal(displacement, initial_velocity));
                    }
                }
            }
        }
    }
}

#endif