#pragma once

#include "pcuditas/gpu/gpu_array.cu"
#include "pcuditas/gpu/gpu_object.cu"
#include "pcuditas/environments/EmptySpace.cu"
#include "force_calculation.cu"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <memory>


class SimpleIntegrator { /*
    * The simplest integrator (Runge-Kutta):
    *   1. update f
    *   2. x -> x + v dt;
    *   3. v -> v + f dt;
    */
    gpu_object<EmptySpace> default_environment;
public:

    SimpleIntegrator() = default;

    template <class ParticleT>
    void operator()(
            gpu_array<ParticleT> &particles,
            double dt = 0.00001) {

        this->update_forces(particles, default_environment);

        this->move(particles, dt);
    }

    template <class ParticleT, class EnvironmentT>
    void operator()(
            gpu_array<ParticleT> &particles, 
            gpu_object<EnvironmentT> &env,
            double dt = 0.001) {

        this->update_forces(particles, env);
        
        this->move(particles, dt);

        // Apply boundary conditions
        particles.for_each(
            [env_ptr=env.gpu_pointer()] 
            __device__ 
            (ParticleT &p, size_t idx) {
                p.position = env_ptr->apply_boundary_conditions(p.position);
            }
        );
    }


    template <class ParticleT, class EnvironmentT>
    void update_forces(
                gpu_array<ParticleT> &particles, 
                gpu_object<EnvironmentT> &env) {
        update_forces_shared(particles, env);
    }

    template <class ParticleT>
    void move(
            gpu_array<ParticleT> &particles,
            double dt = 0.01) {

        particles.for_each(
            [dt] 
            __device__ (ParticleT &p, size_t i) {
                // x -> x + v dt;
                p.position += p.velocity * dt;

                // v -> v + f dt;
                p.velocity += 1/2. * p.force * dt * dt;
            }
        );
    }
};


/* -----------------------------------------------------------------------

 The following is executable documentation as described in Kevlin Henney's talk 
    "Structure and Interpretation of Test Cases" (https://youtu.be/tWn8RA_DEic)
    written using the doctest framework (https://github.com/onqtam/doctest). 

 Run with `make test`.
*/

#ifdef __TESTING__

#include "tests/doctest.h"
#include <typeinfo>   // operator typeid
#include "pcuditas/initial_conditions/random.cu"
#include "pcuditas/particles/SimpleParticle.cu"
#include "pcuditas/vectors/EuclideanVector.cu"

TEST_SUITE("Simple Integrator specification") {

    SCENARIO("No forces") {
        GIVEN("A GPU array of particles with arbitrary positions and velocities") {

            using vector_t = EuclideanVector<3>;
            using particle_t = Particle<vector_t>;
            double L = 100.;
            double V = 10.;

            auto particles = gpu_array<particle_t>(100);
            set_random_positions(particles, L);
            set_random_velocities(particles, V);

            auto snapshot = particles.copy();

            WHEN("A Simple Integrator is used to move them") {
                auto integrator = SimpleIntegrator{};
                int steps = 1000;
                for (int i=0; i<steps; i++) {
                    integrator.move(particles);
                }

                THEN("Each particle has moved along the direction of their velocity") {

                    auto are_colineal = [] (vector_t v1, vector_t v2) {
                        return (
                            (v1.unit_vector() - v2.unit_vector()).magnitude() 
                            == doctest::Approx(0.)
                        );
                    };
                    
                    snapshot.to_cpu();
                    particles.to_cpu();
                    for (int i=0; i<particles.size; i++) {
                        auto initial_velocity = snapshot[i].velocity;
                        auto final_velocity = particles[i].velocity;

                        // Velocity didn't change
                        CHECK(initial_velocity == final_velocity);

                        auto displacement = particles[i].position - snapshot[i].position;

                        CHECK(are_colineal(displacement, initial_velocity));
                    }
                }
            }
        }
    }
}

#endif