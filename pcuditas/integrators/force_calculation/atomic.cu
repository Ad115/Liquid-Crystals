#include "hip/hip_runtime.h"
#pragma once

#include "pcuditas/gpu/gpu_array.cu"

__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

template<class ParticleT, class EnvironmentT>
__global__ 
void update_forces_atomic_kernel(
        ParticleT *particles, int n_particles,
        EnvironmentT *env_ptr) {

    EnvironmentT env = (*env_ptr);

    for (int k = blockIdx.x*blockDim.x + threadIdx.x; 
         k < n_particles*n_particles; 
         k += blockDim.x*gridDim.x) {

        int i = k % n_particles;
        int j = k/n_particles;
        
        double cutoff_radius = 3.5;
        auto dr = env.distance_vector(
            particles[j].position,
            particles[i].position 
        );

        if (dr.magnitude() < cutoff_radius) {
            auto force = ParticleT::force_law(dr);

            for( int d=0; d<force.dimensions; ++d ){
                atomicAddDouble( &(particles[i].force[d]), force[d] );
            }  
        }
    }
}

template<class ParticleT, class EnvironmentT>
void update_forces_atomic(
        gpu_array<ParticleT> &particles,
        gpu_object<EnvironmentT> &environment) {

        using vector_t = typename ParticleT::vector_type;

        // First, reset forces
        particles.for_each([] __device__ (ParticleT& self, int i){
                self.force = vector_t::zero();
        });
        
        // Launch the kernel! As you can see we are not copying memory from CPU to GPU
        // as you would normally do with hipMemcpy(), as we don't need to! The
        // vectors live in GPU already so we just need to know where they start (GPU
        // pointer) and pass it to the kernel.

        unsigned int block_size = 1024;
        unsigned int threads_per_block = 32;
        update_forces_atomic_kernel<<<block_size,threads_per_block>>>(
            particles.gpu_pointer(), particles.size,
            environment.gpu_pointer()
        );
}
