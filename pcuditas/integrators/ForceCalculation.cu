#include "hip/hip_runtime.h"
#pragma once

#include "pcuditas/gpu/gpu_array.cu"


template <typename ParticleT>
__global__ 
void update_forces_shared_kernel(ParticleT *particles, int n_particles) {

    extern __shared__ ParticleT particles_sh[];
    using vector_t = typename ParticleT::vector_type;

    for (int i = blockIdx.x*blockDim.x + threadIdx.x; 
         i < n_particles; 
         i += blockDim.x*gridDim.x) {
        
        auto force = vector_t::zero();
        auto self_pos = particles[i].position;

        // For every other particle
        for (int j=0; j<n_particles; j += blockDim.x) {
            // Copy to shared memory
            particles_sh[threadIdx.x] = particles[j + threadIdx.x];
            __syncthreads();

            // Reduce on block
            for(size_t k=0; k<blockDim.x; k++) {
                auto other_pos = particles[k].position;
                auto dr = self_pos - other_pos;

                auto f_ij = (i != k) ? ParticleT::force_law(dr) : vector_t::zero();
                force += f_ij;   
            }
            __syncthreads();
        }

        // Save the results
        particles[i].force = force;
    }
}

template<class ParticleT>
void update_forces_shared(
            gpu_array<ParticleT> &particles,
            unsigned int block_size = 1024,
            unsigned int threads_per_block = 32) {
        
        // Launch the kernel. As you can see we are not copying memory from CPU to GPU
        // as you would normally do with hipMemcpy(), as we don't need to! The
        // vectors live in GPU already so we just need to know where they start (GPU
        // pointer) and pass it to the kernel.

        unsigned int shared_memory_size = threads_per_block * sizeof(ParticleT);
        update_forces_shared_kernel<<<block_size, threads_per_block, shared_memory_size>>>(
            particles.gpu_pointer(), particles.size
        );
}


// ===   ===   ===

template<class ParticleT>
void update_forces_naive(gpu_array<ParticleT> &particles) {

        using vector_t = typename ParticleT::vector_type;

        // Naïve paralellization.
        particles.for_each(
            [others=particles.gpu_pointer(), n=particles.size] 
            __device__ (ParticleT& self, int i) {
                auto force = vector_t::zero();
                auto self_pos = self.position;

                for(int j=0; j<n; j++) {
                    auto other_pos = others[j].position;
                    auto dr = self_pos - other_pos;

                    auto f_ij = (i != j) ? ParticleT::force_law(dr) : vector_t::zero();
                    force += f_ij;    
                }

                self.force = force;
        });
}


// ===  ===  === 


__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long int* address_as_ull =
                             (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

template <typename ParticleT>
__global__ 
void update_forces_atomic_kernel(ParticleT *particles, int n_particles) {

    for (int k = blockIdx.x*blockDim.x + threadIdx.x; 
         k < n_particles*n_particles; 
         k += blockDim.x*gridDim.x) {

        int i = k % n_particles;
        int j = k/n_particles;
        
        double cutoff_radius = 3.5;
        auto dr = (
            particles[i].position - particles[j].position
        );

        if (dr.magnitude() < cutoff_radius) {
            auto force = ParticleT::force_law(dr);

            for( int d=0; d<force.dimensions; ++d ){
                atomicAddDouble( &(particles[i].force[d]), force[d] );
            }  
        }
    }
}

template<class ParticleT>
void update_forces_atomic(gpu_array<ParticleT> &particles) {

        using vector_t = typename ParticleT::vector_type;

        // First, reset forces
        particles.for_each([] __device__ (ParticleT& self, int i){
                self.force = vector_t::zero();
        });
        
        // Launch the kernel! As you can see we are not copying memory from CPU to GPU
        // as you would normally do with hipMemcpy(), as we don't need to! The
        // vectors live in GPU already so we just need to know where they start (GPU
        // pointer) and pass it to the kernel.

        unsigned int block_size = 1024;
        unsigned int threads_per_block = 32;
        update_forces_atomic_kernel<<<block_size,threads_per_block>>>(
            particles.gpu_pointer(), particles.size
        );
}

