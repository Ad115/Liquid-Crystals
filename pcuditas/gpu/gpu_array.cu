#include "hip/hip_runtime.h"
#pragma once

/* 
## Clase `gpu_array`

Un *smart pointer* para arreglos de objetos en el GPU. La contraparte del 
`gpu_object` pero para arreglos.

La clase abstrae la alocación y liberación de memoria además de las operaciones 
de copia entre Host y Device. Esta es una abstracción del Host, por lo que no 
se puede utilizar en un kernel. 
*/

#include "pcuditas/gpu/macros.cu"
#include "pcuditas/gpu/kernels.cu"
#include "pcuditas/gpu/gpu_object.cu"
#include <assert.h>



template< typename T >
class gpu_array {
  
    T *_gpu_pointer;
    T *_cpu_pointer;
    
    public:

    size_t size;
    using element_t = T;
    
    gpu_array(size_t n): size(n) {
        assert(size > 0);

        // <-- Allocate and initialize on GPU
        CUDA_CALL(hipMalloc(&_gpu_pointer, n * sizeof(T)));      

        init_array_kernel<<<128,32>>>(_gpu_pointer, n);


        // <-- Allocate and initialize on CPU
        _cpu_pointer = (T *) malloc(n * sizeof(T));

        for (int i=0; i<n; i++) {
            new (&_cpu_pointer[i]) T();
        }
    }

    template <class InitializerT>
    gpu_array(size_t n, InitializerT init_fn)
            : gpu_array(n) { /*

        Instantiate with a function to initialize each value.

        Example:
            // Initialize to {0, 1, 2, 3, 4...9}
            auto array = gpu_array<int>(10, 
                []__device__ (int &el, int i) {
                    el = i;
            });
        */

        // Apply the initialization function to each element
        (*this).for_each(init_fn);
    }
    
    T *gpu_pointer() const {
        return _gpu_pointer;
    }
    
    T *to_cpu() { 
        CUDA_CALL(hipMemcpy(
            _cpu_pointer, _gpu_pointer, 
            size*sizeof(T), 
            hipMemcpyDeviceToHost
        ));

        return _cpu_pointer;
    }

    T *cpu_pointer() const {
        return _cpu_pointer;
    }

    T operator[](size_t idx) {
        return _cpu_pointer[idx];
    }

    template <class TransformedT, class TransformationT>
    gpu_array<TransformedT> transform(
            TransformationT gpu_fn,
            int n_blocks = 1024, 
            int n_threads = 32 ) { /*

        Create a new array with the transformed elements.
        
        Example:

            // Create gpu_array "array" with the numbers 0 to 9
            auto array = gpu_array<int>(10, 
                []__device__ (int &el, int i) {
                    el = i;
            });
            
            
            // Transform to pairs of the number and it's squares
            auto squares = array.transform<int2>(
                [] __device__ (int2 el, int idx) {
                    return make_int2(el, el*el);
            });
    */

        auto transformed = gpu_array<TransformedT>{this->size};

        transform_kernel<<<n_blocks, n_threads>>>(
            _gpu_pointer, size, 
            transformed.gpu_pointer(),
            gpu_fn
        );

        return transformed;
    }

    template <class FunctionT>
    gpu_array<T>& for_each(
            FunctionT gpu_fn,
            int n_blocks = 1024, 
            int n_threads = 32 ) {/*

        Apply the function in-place for each element of the array.
        
        Example:

            // Create gpu_array "array" with the numbers 0 to 9
            auto array = gpu_array<int>(10, 
                []__device__ (int &el, int i) {
                    el = i;
            });
            
            
            // Make a linear transformation
            int a = 12;
            int b = 34;

            array.for_each(
                [a,b] __device__ (int2 &el, int idx) {
                    el = a*el + b;
            });
        */

        for_each_kernel<<<n_blocks, n_threads>>>(_gpu_pointer, size, gpu_fn);
        return *this;
    }

    template <class ReductionT>
    gpu_object<T> reduce(
            ReductionT reduce_fn, 
            int n_blocks = 128, 
            int threads_per_block = 32 /* <-- Must be a power of 2! */ ) { /*
        
        Perform a reduction of the elements of the array using the provided function.

        Example:

            // Create gpu_array "array" with the numbers 0 to 9
            auto array = gpu_array<int>(10, 
                []__device__ (int &el, int i) {
                    el = i;
            });
            
            
            // Multipliy the elements in GPU
            gpu_object<int> product 
                = array.reduce(
                    []__device__ (int a, int b) {
                        return a * b
                });
        */

        unsigned int shared_memory_size = threads_per_block * sizeof(T);

        auto block_partials = gpu_array<T>(n_blocks);
        reduce_2step_kernel<<<n_blocks, threads_per_block, shared_memory_size>>>(
            _gpu_pointer, size, block_partials.gpu_pointer(), reduce_fn
        );

        auto out = gpu_object<T>();
        reduce_2step_kernel<<<1, threads_per_block, shared_memory_size>>>(
            block_partials.gpu_pointer(), n_blocks, out.gpu_pointer(), reduce_fn
        );

        return out;
    }

    // Iterator protocol
    T* begin() { return _cpu_pointer; }
    T* end() { return _cpu_pointer + size; }
    
    gpu_array<T> copy() {
        auto copied = gpu_array<T>(this->size);

        // Copy in GPU
        copied.for_each(
            [old_one=this->gpu_pointer()]
            __device__ (T &new_el, int i) {
                new_el = old_one[i];
        });

        // Copy in CPU
        for(int i=0; i<size; i++) {
            copied[i] = (*this)[i];
        }

        return copied;
    }

    ~gpu_array() {
        free(_cpu_pointer);
        CUDA_CALL(hipFree(_gpu_pointer));
    }
};



/* -----------------------------------------------------------------------

 The following is executable documentation as described in Kevlin Henney's talk 
    "Structure and Interpretation of Test Cases" (https://youtu.be/tWn8RA_DEic)
    written using the doctest framework (https://github.com/onqtam/doctest). 

 Run with `make test`.
*/

#ifdef __TESTING__

#include "tests/doctest.h"
#include <typeinfo>   // operator typeid
#include <assert.h>

template<class T>
struct pair {
    T first; T second;
};

TEST_SUITE("GPU Array specification") {

    SCENARIO("GPU Array initialization") {

        GIVEN("A size and the type of the elements") {

            int size = 10;
            using element_t = int;

            THEN("A GPU array can be initialized without failure") {

                auto array = gpu_array<element_t>(size);

                using array_element_t = decltype(array)::element_t;
                CHECK(typeid(array_element_t) == typeid(element_t));
                CHECK(array.size == size);
            }
        }
    }

    SCENARIO("GPU Array for_each") {
        GIVEN("A GPU array") {
            int size = 10;
            using element_t = int;

            auto array = gpu_array<element_t>(size);

            WHEN("It's elements are modified with for_each") {
                array.for_each(
                    [] __device__ (element_t &el, int idx) {
                        el = idx * idx;
                });

                THEN("The values on GPU are changed accordingly") {

                    array.for_each( // <-- check on GPU
                        [] __device__ (element_t current_val, int idx){
                            assert(current_val == idx*idx);
                    });

                    array.to_cpu(); // <-- check on CPU
                    for(int i=0; i<array.size; i++){
                        CHECK(array[i] == i*i);
                    }
                }
            }
        }
    }

    SCENARIO("GPU Array transformation") {
        GIVEN("A GPU array") {
            int size = 10;
            using element_t = int;

            // Initialize to {0, 1, 2, 3, 4...9}
            auto array = gpu_array<element_t>(size, 
                []__device__ (element_t &el, int i) {
                    el = i;
            });

            WHEN("A new array is obtained as a transformation of it") {

                auto squares = array.transform<pair<element_t>>(
                    [] __device__ (element_t &el, int idx) {
                        return pair<element_t>{el, el*el};
                });

                THEN("The values on GPU are changed accordingly") {

                    squares.for_each( // <-- check on GPU
                        [] __device__ (pair<element_t> p, int idx) {
                            assert(p.first == idx);
                            assert(p.second == idx*idx);
                    });

                    squares.to_cpu(); // <-- check on CPU
                    for(int i=0; i<squares.size; i++){
                        CHECK(squares[i].first == i);
                        CHECK(squares[i].second == i*i);
                    }
                }
            }
        }
    }

    SCENARIO("GPU Array reduction") {
        GIVEN("A GPU array with arbitrary elements") {
            int size = 1000;
            using element_t = int;

            auto array = gpu_array<element_t>(size);
            array.for_each(
                [] __device__ (element_t &el, int i) {
                    el = i+1;
            });

            WHEN("A reducion operation is applied on it") {
                auto sum_gpu = array.reduce(
                    [] __device__ (element_t reduced, element_t el) {
                        return reduced + el;
                }).to_cpu();

                THEN("The reduction on CPU yields the same result") {

                    array.to_cpu(); // <-- check on CPU

                    auto sum_cpu = array[0];
                    for(int i=1; i<array.size; i++){
                        sum_cpu += array[i];
                    }

                    CHECK(sum_cpu == sum_gpu);
                }
            }

        }

        SUBCASE("Reduction on a very large array") {
            auto n = 500000;
            
            // Initialize to {0, 1, 2, 3, 4...n}
            auto nums = gpu_array<int>(n, 
                [] __device__ (int &el, int idx) {
                    el = idx;
            });

            auto addition = 
                [] __device__ (int a, int b) {
                    return a + b;
            };

            auto sum = nums.reduce(addition).to_cpu();

            CHECK(sum == n*(n-1)/2);
        }
    }
}
#endif