#include "hip/hip_runtime.h"
#pragma once

/* 
## Clase `gpu_array`

Un *smart pointer* para arreglos de objetos en el GPU. La contraparte del 
`gpu_object` pero para arreglos.

La clase abstrae la alocación y liberación de memoria además de las operaciones 
de copia entre Host y Device. Esta es una abstracción del Host, por lo que no 
se puede utilizar en un kernel. 
*/

#include "pcuditas/gpu/macros.cu"
#include "pcuditas/gpu/gpu_object.cu"


template<typename T>
__global__
void _init_array_kernel(T *gpu_array, size_t n) {

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
        i < n; 
        i += blockDim.x * gridDim.x) 
    {
        new (&gpu_array[i]) T();
    }
}

template<class T, class TransformedT, class TransformationT>
__global__
void _transform_kernel(
        T *from_array, size_t n, 
        TransformedT *to_array,
        TransformationT transform) {

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
        i < n; 
        i += blockDim.x * gridDim.x) 
    {
        to_array[i] = transform(from_array[i], i);
    }
}

template<typename T, typename Transformation>
__global__
void _for_each_kernel(T *gpu_array, size_t n, Transformation fn) {

    for (int i = blockIdx.x * blockDim.x + threadIdx.x; 
        i < n; 
        i += blockDim.x * gridDim.x) 
    {
        fn(gpu_array[i], i);
    }
}

template<typename T, typename Reduction>
__global__
void _reduce_kernel(
        T *gpu_array, size_t n, 
        T *out, 
        Reduction fn, T initial_value=T{}) { /*
    Log-reduction based from the one in the book "The CUDA Handbook" by 
    Nicholas Wilt.
    */

    extern __shared__ T partials[];

    const int tid = threadIdx.x;

    auto reduced = initial_value;
    for (int i = blockIdx.x * blockDim.x + tid; 
         i < n; 
         i += blockDim.x * gridDim.x) {

        reduced = fn(reduced, gpu_array[i]);
    }
    partials[tid] = reduced;
    __syncthreads();


    for (int active_threads = blockDim.x / 2;
         active_threads > 0;
         active_threads /= 2) {
        
        auto is_active_thread = tid < active_threads;
        if (is_active_thread) {
            partials[tid] = fn(partials[tid], partials[tid + active_threads]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        out[blockIdx.x] = partials[0];
    }
}



template< typename T >
class gpu_array {
  
    T *_gpu_pointer;
    T *_cpu_pointer;
    
    public:

    size_t size;
    using element_t = T;
    
    gpu_array(size_t n): size(n) {
        // <-- Allocate and initialize on GPU
        CUDA_CALL(hipMalloc(&_gpu_pointer, n * sizeof(T)));      

        _init_array_kernel<<<128,32>>>(_gpu_pointer, n);


        // <-- Allocate and initialize on CPU
        _cpu_pointer = (T *) malloc(n * sizeof(T));

        for (int i=0; i<n; i++) {
            new (&_cpu_pointer[i]) T();
        }
    }

    // Instantiate with a function to initialize each value
    template <class InitializerT>
    gpu_array(size_t n, InitializerT init_fn)
        : gpu_array(n) { 
        // Apply the initialization function to each element
        (*this).for_each(init_fn);
    }
    
    T *gpu_pointer() const {
        return _gpu_pointer;
    }
    
    T *to_cpu() {
        CUDA_CALL(hipMemcpy(
            _cpu_pointer, _gpu_pointer, 
            size*sizeof(T), 
            hipMemcpyDeviceToHost
        ));

        return _cpu_pointer;
    }

    T *cpu_pointer() const {
        return _cpu_pointer;
    }

    T operator[](size_t idx) {
        return _cpu_pointer[idx];
    }

    template <class TransformedT, class TransformationT>
    gpu_array<TransformedT> transform(
            TransformationT gpu_fn,
            int n_blocks = 1024, 
            int n_threads = 32 ){

        auto transformed = gpu_array<TransformedT>{this->size};

        _transform_kernel<<<n_blocks, n_threads>>>(
            _gpu_pointer, size, 
            transformed.gpu_pointer(),
            gpu_fn
        );

        return transformed;
    }

    template <class FunctionT>
    gpu_array<T>& for_each(
            FunctionT gpu_fn,
            int n_blocks = 1024, 
            int n_threads = 32 ) {
        _for_each_kernel<<<n_blocks, n_threads>>>(_gpu_pointer, size, gpu_fn);
        return *this;
    }

    template <class ReductionT>
    gpu_object<T> reduce(
            ReductionT reduce_fn, 
            int n_blocks = 128, 
            int threads_per_block = 32 /* <-- Must be a power of 2! */ ) {

        unsigned int shared_memory_size = threads_per_block * sizeof(T);

        auto thread_partials = gpu_array<T>(threads_per_block);
        _reduce_kernel<<<n_blocks, threads_per_block, shared_memory_size>>>(
            _gpu_pointer, size, thread_partials.gpu_pointer(), reduce_fn
        );

        auto out = gpu_object<T>();
        _reduce_kernel<<<1, threads_per_block, shared_memory_size>>>(
           thread_partials.gpu_pointer(), size, out.gpu_pointer(), reduce_fn
        );

        return out;
    }

    // Iterator protocol
    T* begin() { return _cpu_pointer; }
    T* end() { return _cpu_pointer + size; }
    
    gpu_array<T> copy() {
        auto copied = gpu_array<T>(this->size);

        // Copy in GPU
        copied.for_each(
            [old_one=this->gpu_pointer()]
            __device__ (T &new_el, int i) {
                new_el = old_one[i];
        });

        // Copy in CPU
        for(int i=0; i<size; i++) {
            copied[i] = (*this)[i];
        }

        return copied;
    }

    ~gpu_array() {
        free(_cpu_pointer);
        CUDA_CALL(hipFree(_gpu_pointer));
    }
};



/* -----------------------------------------------------------------------

 The following is executable documentation as described in Kevlin Henney's talk 
    "Structure and Interpretation of Test Cases" (https://youtu.be/tWn8RA_DEic)
    written using the doctest framework (https://github.com/onqtam/doctest). 

 Run with `make test`.
*/

#ifdef __TESTING__

#include "tests/doctest.h"
#include <typeinfo>   // operator typeid
#include <assert.h>

template<class T>
struct pair {
    T first; T second;
};

TEST_SUITE("GPU Array specification") {

    SCENARIO("GPU Array initialization") {

        GIVEN("A size and the type of the elements") {

            int size = 10;
            using element_t = int;

            THEN("A GPU array can be initialized without failure") {

                auto array = gpu_array<element_t>(size);

                using array_element_t = decltype(array)::element_t;
                CHECK(typeid(array_element_t) == typeid(element_t));
                CHECK(array.size == size);
            }
        }
    }

    SCENARIO("GPU Array for_each") {
        GIVEN("A GPU array") {
            int size = 10;
            using element_t = int;

            auto array = gpu_array<element_t>(size);

            WHEN("It's elements are modified with for_each") {
                array.for_each(
                    [] __device__ (element_t &el, int idx) {
                        el = idx * idx;
                });

                THEN("The values on GPU are changed accordingly") {

                    array.for_each( // <-- check on GPU
                        [] __device__ (element_t current_val, int idx){
                            assert(current_val == idx*idx);
                    });

                    array.to_cpu(); // <-- check on CPU
                    for(int i=0; i<array.size; i++){
                        CHECK(array[i] == i*i);
                    }
                }
            }
        }
    }

    SCENARIO("GPU Array transformation") {
        GIVEN("A GPU array") {
            int size = 10;
            using element_t = int;

            // Initialize to {0, 1, 2, 3, 4...9}
            auto array = gpu_array<element_t>(size, 
                []__device__ (element_t &el, int i) {
                    el = i;
            });

            WHEN("A new array is obtained as a transformation of it") {

                auto squares = array.transform<pair<element_t>>(
                    [] __device__ (element_t &el, int idx) {
                        return pair<element_t>{el, el*el};
                });

                THEN("The values on GPU are changed accordingly") {

                    squares.for_each( // <-- check on GPU
                        [] __device__ (pair<element_t> p, int idx) {
                            assert(p.first == idx);
                            assert(p.second == idx*idx);
                    });

                    squares.to_cpu(); // <-- check on CPU
                    for(int i=0; i<squares.size; i++){
                        CHECK(squares[i].first == i);
                        CHECK(squares[i].second == i*i);
                    }
                }
            }
        }
    }

    SCENARIO("GPU Array reduction") {
        GIVEN("A GPU array with arbitrary elements") {
            int size = 1000;
            using element_t = int;

            auto array = gpu_array<element_t>(size);
            array.for_each(
                [] __device__ (element_t &el, int i) {
                    el = i+1;
            });

            WHEN("A reducion operation is applied on it") {
                auto sum_gpu = array.reduce(
                    [] __device__ (element_t reduced, element_t el) {
                        return reduced + el;
                }).to_cpu();

                THEN("The reduction on CPU yields the same result") {

                    array.to_cpu(); // <-- check on CPU

                    auto sum_cpu = array[0];
                    for(int i=1; i<array.size; i++){
                        sum_cpu += array[i];
                    }

                    CHECK(sum_cpu == sum_gpu);
                }
            }

        }
    }
}
#endif