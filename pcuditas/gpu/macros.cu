#pragma once
#include <hip/hip_runtime.h>


#define CUDA_CALL(x) do { if((x) != hipSuccess) { \
    printf("Error at %s:%d -- %s\n",__FILE__,__LINE__, hipGetErrorString(x));}} while(0)
